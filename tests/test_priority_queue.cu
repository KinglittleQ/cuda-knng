#include "hip/hip_runtime.h"
#include <iostream>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include <knng/config.cuh>
#include <knng/priority_queue.cuh>
#include <knng/utils.cuh>

__global__ void TestPriorityQueue() {
  knng::PriorityQueue queue(32);
  for (int i = 0; i < 100; i++) {
    queue.Add(i, 100.0f - i);
    queue.Add(i, 100.0f - i);
  }
  queue.Print();
  if (threadIdx.x == 0) {
    printf("top: %d\n", queue.ids[0]);
  }
}

int main(void) {
  TestPriorityQueue<<<1, BLOCK_DIM_X>>>();
  CheckCudaStatus();

  return 0;
}
