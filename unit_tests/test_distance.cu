#include "hip/hip_runtime.h"
#include <iostream>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include <knng/config.cuh>
#include <knng/utils.cuh>
#include <knng/distance.cuh>


__global__ void ComputeDistance(float *data, uint32_t a, uint32_t b, float *result) {
  L2Distance distance(data, data + a * DIM);
  float ret = distance.Compare(b);
  if (threadIdx.x == 0) {
    *result = ret;
  }
}


int main(void) {
  float data[2 * DIM];
  float result;
  float *cuda_data, *cuda_result;

  for (int i = 0; i < DIM; i++) {
    data[i] = 1.0f;
    data[i + DIM] = 3.0f;
  }

  hipMalloc(&cuda_data, 2 * DIM * sizeof(float));
  hipMemcpy(cuda_data, data, 2 * DIM * sizeof(float), hipMemcpyHostToDevice);
  hipMalloc(&cuda_result, sizeof(float));

  ComputeDistance<<<1, BLOCK_DIM_X>>>(cuda_data, 0, 1, cuda_result);
  CheckCudaStatus();

  hipMemcpy(&result, cuda_result,sizeof(float), hipMemcpyDeviceToHost);
  std::cout << result << std::endl;

  hipFree(cuda_data);
  hipFree(cuda_result);

  return 0;
}